﻿#include "kernel.cuh"

const char* matrixToString(const float* A, int m, int n)
{
    std::ostringstream oss;
    oss << "Matrix:\n";
    oss << std::fixed << std::setprecision(5);
    for (int i = 0; i < m; ++i) {
        for (int j = 0; j < n; ++j) {
            float value = A[i + m * j];
            oss << value << " ";
        }
        oss << "\n";
    }
    std::string result = oss.str();
    return result.c_str();
}

void matrixMultiplyGPU(const float* A, const float* B, float* C, int m, int n, int k) {
    hipError_t cudaStatus;
    hipblasStatus_t cublasStat;
    float *d_A, *d_B, *d_C;
    hipblasHandle_t handle;

    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed!\n");
        goto Error;
    }

    cublasStat = hipblasCreate(&handle);
    if (cublasStat != HIPBLAS_STATUS_SUCCESS) {
        fprintf(stderr, "CUBLAS initialization failed\n");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&d_A, m * k * sizeof(float));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!\n");
        goto Error;
    }
    cudaStatus = hipMalloc((void**)&d_B, k * n * sizeof(float));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!\n");
        goto Error;
    }
    cudaStatus = hipMalloc((void**)&d_C, m * n * sizeof(float));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!\n");
        goto Error;
    }

    cublasStat = hipblasSetMatrix(m, k, sizeof(float), A, m, d_A, m);
    if (cublasStat != HIPBLAS_STATUS_SUCCESS) {
        fprintf(stderr, "CUBLAS set matrix failed\n");
        goto Error;
    }
    cublasStat = hipblasSetMatrix(k, n, sizeof(float), B, k, d_B, k);
    if (cublasStat != HIPBLAS_STATUS_SUCCESS) {
        fprintf(stderr, "CUBLAS set matrix failed\n");
        goto Error;
    }

    // Perform matrix multiplication: C = alpha * A * B + beta * C
    const float alpha = 1.0f;
    const float beta = 0.0f;
    cublasStat = hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, m, n, k, &alpha, d_A, m, d_B, k, &beta, d_C, m);
    if (cublasStat != HIPBLAS_STATUS_SUCCESS) {
        fprintf(stderr, "CUBLAS matrix multiplication failed\n");
        goto Error;
    }

    cublasStat = hipblasGetMatrix(m, n, sizeof(float), d_C, m, C, m);
    if (cublasStat != HIPBLAS_STATUS_SUCCESS) {
        fprintf(stderr, "CUBLAS get matrix failed\n");
        goto Error;
    }
    hipDeviceSynchronize();

Error:
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
    hipblasDestroy(handle);
}

void matrixMultiplyCPU(const float* A, const float* B, float* C, int m, int n, int k)
{
    for (int i = 0; i < m * n; ++i) {
        C[i] = 0.0f;
    }

    for (int j = 0; j < n; ++j) {
        for (int i = 0; i < m; ++i) {
            for (int p = 0; p < k; ++p) {
                C[i + j * m] += A[i + p * m] * B[p + j * k];
            }
        }
    }
}
